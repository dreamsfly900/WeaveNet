#ifndef __CONV2D3X3_H__
#define __CONV2D3X3_H__
#include <hip/hip_runtime.h>

extern void conv2Mex(float* in, float* out, int numRows, int numCols, float* kernel);

#endif // __CONV2D3X3_H__

extern "C"
__global__ void conv2MexCuda(float* src,
    float* dst,
    int numRows,
    int numCols,
    float* kernel)
{
    int row = blockIdx.x;
    if (row < 1 || row > numRows - 1)
        return;

    int col = blockIdx.y;
    if (col < 1 || col > numCols - 1)
        return;

    int dstIndex = col * numRows + row;
    dst[dstIndex] = 0;
    int kerIndex = 3 * 3 - 1;
    for (int kc = -1; kc < 2; kc++)
    {
        int srcIndex = (col + kc) * numRows + row;
        for (int kr = -1; kr < 2; kr++)
        {
            dst[dstIndex] += kernel[kerIndex--] * src[srcIndex + kr];
        }
    }
}

void conv2Mex(float* src, float* dst, int numRows, int numCols, float* ker)
{
    int totalPixels = numRows * numCols;
    float* deviceSrc, * deviceKer, * deviceDst;

    hipMalloc(&deviceSrc, sizeof(float) * totalPixels);
    hipMalloc(&deviceDst, sizeof(float) * totalPixels);
    hipMalloc(&deviceKer, sizeof(float) * 3 * 3);

    hipMemcpy(deviceSrc, src, sizeof(float) * totalPixels, hipMemcpyHostToDevice);
    hipMemcpy(deviceKer, ker, sizeof(float) * 3 * 3, hipMemcpyHostToDevice);
    hipMemset(deviceDst, 0, sizeof(float) * totalPixels);

    dim3 gridSize(numRows, numCols);
    conv2MexCuda<<<gridSize,1>>>(deviceSrc, deviceDst, numRows, numCols, deviceKer);

    hipMemcpy(dst, deviceDst, sizeof(float) * totalPixels, hipMemcpyDeviceToHost);

    hipFree(deviceSrc);
    hipFree(deviceDst);
    hipFree(deviceKer);
}