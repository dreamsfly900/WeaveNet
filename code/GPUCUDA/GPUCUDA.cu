﻿#include "hip/device_functions.h"
#include "hip/hip_runtime.h"

#include "stdlib.h"
#include <iostream>
//#include<stdio.h>
using namespace std;
//返回thread和block
int getThreadNum()
{
	hipDeviceProp_t prop;//cudaDeviceProp的一个对象
	int count = 0;//GPU的个数
	hipGetDeviceCount(&count);
	std::cout << "gpu 的个数：" << count << '\n';

	hipGetDeviceProperties(&prop, 0);//第二参数为那个gpu
	cout << "最大线程数：" << prop.maxThreadsPerBlock << endl;
	cout << "最大网格类型：" << prop.maxGridSize[0] << '\t' << prop.maxGridSize[1] << '\t' << prop.maxGridSize[2] << endl;
	return prop.maxThreadsPerBlock;
}
__global__ void conv(float* imgGpu, float* kernelGpu, float* resultGpu, int width, int height, int kernelSize)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id >= width * height)
	{
		return;
	}
	int row = id / width;//获取img 的行和列
	int clo = id % width;
	//每一个线程处理一次卷积计算
	//resultGpu[id] = 0;
	for (int i = 0; i < kernelSize; ++i)
	{
		for (int j = 0; j < kernelSize; ++j)
		{
			float imgValue = 0;//记录结果
			//imgValue += kernelGpu[i*kernelSize + j] * imgGpu[id];
			int curRow = row - kernelSize / 2 + i;
			int curClo = clo - kernelSize / 2 + j;
			if (curRow < 0 || curClo < 0 || curRow >= height || curClo >= width)
			{
			}
			else
			{
				//imgValue += kernelGpu[i*kernelSize + j] * imgGpu[(curRow + i - 1)*width + curClo + j - 1];
				imgValue = imgGpu[curRow * width + curClo];

			}
			resultGpu[id] += kernelGpu[i * kernelSize + j] * imgValue;

		}
	}
}

//形参：枚举类型
void GetCudaCalError(hipError_t err)
{
	if (err != hipSuccess)
	{
		cout << "分配内存失败！程序结束！";
	}
	return;
}
int main()
{
	//定义一个1080p照片
	const int width = 1920;
	const int height = 1080;
	//float *img = (float*)calloc(width*height, sizeof(float));
	float* img = new float[width * height];
	//赋值
	for (int row = 0; row < height; ++row)
	{
		for (int col = 0; col < width; ++col)
		{
			img[col + row * width] = (col + row) % 256;
		}
	}
	//声明卷积核大小,大小为3*3
	const int kernelSize = 3;
	//float*kernel = (float*)calloc(kernelSize*kernelSize, sizeof(float));
	float* kernel = new float[kernelSize * kernelSize];
	//卷积核赋值
	//第一种方法
	for (int i = 0; i < kernelSize; ++i)
	{
		for (int j = 0; j < kernelSize; ++j)
		{
			kernel[i + j * kernelSize] = i - 1;
		}
	}
	//第二种
	/*for (int i = 0; i < kernelSize*kernelSize; ++i)
	{
		kernel[i] = i % kernelSize - 1;
	}*/
	//输出img的左上角
	for (int row = 0; row < 10; ++row)
	{
		for (int col = 0; col < 10; ++col)
		{
			std::cout << img[col + row * width] << '\t';
		}
		std::cout << '\n';
	}
	cout << "kernel\n";
	for (int i = 0; i < kernelSize; ++i)
	{
		for (int j = 0; j < kernelSize; ++j)
		{
			std::cout << kernel[i * kernelSize + j] << '\t';
		}
		cout << endl;

	}


	float* imgGpu = 0;//将host值复制到device上面
	float* kernelGpu = 0;//将kernel也复制到device上
	float* resultGpu = 0;//卷积结果

	//为Device分配内存
	GetCudaCalError(hipMalloc(&imgGpu, height * width * sizeof(float)));
	GetCudaCalError(hipMalloc(&kernelGpu, kernelSize * kernelSize * sizeof(float)));
	GetCudaCalError(hipMalloc(&resultGpu, height * width * sizeof(float)));
	//这个地方捕捉错误，明天改

	hipMemcpy(imgGpu, img, width * height * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(kernelGpu, kernel, kernelSize * kernelSize * sizeof(float), hipMemcpyHostToDevice);
	//获取GPU信息
	const int threadNum = getThreadNum();
	const int blockNum = (width * height + threadNum - 1) / threadNum;//这里block使用一维
	//conv(imgGpu, kernelGpu, resultGpu, width, height, kernelSize);
    conv <<<blockNum, threadNum >>> (imgGpu, kernelGpu, resultGpu, width, height, kernelSize);
	//接受Device上resultGpu里面的数据
	float* showImg = new float[height * width];
	hipMemcpy(showImg, resultGpu, width * height * sizeof(float), hipMemcpyDeviceToHost);

	for (int row = 0; row < 10; ++row)
	{
		for (int col = 0; col < 10; ++col)
		{
			std::cout << showImg[col + row * width] << '\t';
		}
		std::cout << '\n';
	}
	//没有释放内存
	hipFree(imgGpu);
	hipFree(kernelGpu);
	hipFree(resultGpu);
	/*free(img);
	free(kernel);*/
	delete[] img;
	delete[] kernel;
	delete[] showImg;
	system("pause");
	return 0;
}